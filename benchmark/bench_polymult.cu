#include <cstdlib>  // For atoi or atof functions
#include <iomanip>
#include <iostream>
#include <random>

#include "fft.cuh"
#include "fft_cpu.cuh"

using namespace std;
using namespace fft;

int q;
int logn;
int batch;
int n;

int main(int argc, char* argv[])
{
    if(argc < 3)
    {
        q = 7;
        logn = 11;
        batch = 1;
        n = 1 << logn;
    }
    else
    {
        q = 7;
        logn = atoi(argv[1]);
        batch = atoi(argv[2]);
        n = 1 << logn;

        if((logn < 12) || (24 < logn))
        {
            throw std::runtime_error("LOGN should be in range 12 to 24.");
        }
    }

    const int test_count = 50;
    const int bestof = 10;
    float time_measurements[test_count];
    for(int loop = 0; loop < test_count; loop++)
    {
        std::random_device rd;
        std::mt19937 gen(rd());
        // std::mt19937 gen(0);
        unsigned long long minNumber = 0;
        unsigned long long maxNumber = q - 1;
        std::uniform_int_distribution<int> dis(minNumber, maxNumber);

        std::vector<std::vector<COMPLEX_C>> vec_GPU(2 * batch, std::vector<COMPLEX_C>(n * 2));  // A and B together

        for(int j = 0; j < 2 * batch; j++)
        {  // LOAD A
            for(int i = 0; i < n * 2; i++)
            {
                COMPLEX_C element = dis(gen);
                vec_GPU[j][i] = element;
            }
        }

        FFT fft_generator(n);

        /////////////////////////////////////////////////////////////////////////

        COMPLEX* Forward_InOut_Datas;

        THROW_IF_CUDA_ERROR(hipMalloc(&Forward_InOut_Datas,
                                       2 * batch * n * 2 * sizeof(COMPLEX)));  // 2 --> A and B, batch --> batch
                                                                               // size, 2 --> zero pad

        for(int j = 0; j < 2 * batch; j++)
        {
            THROW_IF_CUDA_ERROR(hipMemcpy(Forward_InOut_Datas + (n * 2 * j), vec_GPU[j].data(),
                                           n * 2 * sizeof(COMPLEX), hipMemcpyHostToDevice));
        }
        /////////////////////////////////////////////////////////////////////////

        COMPLEX* Root_Table_Device;

        THROW_IF_CUDA_ERROR(hipMalloc(&Root_Table_Device, n * sizeof(COMPLEX)));

        vector<COMPLEX_C> reverse_table = fft_generator.ReverseRootTable();
        THROW_IF_CUDA_ERROR(
            hipMemcpy(Root_Table_Device, reverse_table.data(), n * sizeof(COMPLEX), hipMemcpyHostToDevice));

        /////////////////////////////////////////////////////////////////////////

        COMPLEX* Inverse_Root_Table_Device;

        THROW_IF_CUDA_ERROR(hipMalloc(&Inverse_Root_Table_Device, n * sizeof(COMPLEX)));

        vector<COMPLEX_C> inverse_reverse_table = fft_generator.InverseReverseRootTable();
        THROW_IF_CUDA_ERROR(hipMemcpy(Inverse_Root_Table_Device, inverse_reverse_table.data(), n * sizeof(COMPLEX),
                                       hipMemcpyHostToDevice));

        /////////////////////////////////////////////////////////////////////////

        unsigned long long* activity_output;
        THROW_IF_CUDA_ERROR(hipMalloc(&activity_output, 64 * 512 * sizeof(unsigned long long)));
        GPU_ACTIVITY_HOST(activity_output, 111111);
        THROW_IF_CUDA_ERROR(hipFree(activity_output));

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        hipDeviceSynchronize();

        fft_configuration cfg_fft = {.n_power = (logn + 1), .ntt_type = FORWARD, .zero_padding = false, .stream = 0};
        fft_configuration cfg_ifft = {.n_power = (logn + 1),
                                      .ntt_type = INVERSE,
                                      .zero_padding = false,
                                      .mod_inverse = COMPLEX(fft_generator.n_inverse, 0.0),
                                      .stream = 0};

        float time = 0;
        hipEvent_t startx, stopx;
        hipEventCreate(&startx);
        hipEventCreate(&stopx);

        hipEventRecord(startx);

        GPU_FFT(Forward_InOut_Datas, Root_Table_Device, cfg_fft, batch * 2, false);

        GPU_FFT(Forward_InOut_Datas, Inverse_Root_Table_Device, cfg_ifft, batch, true);

        hipEventRecord(stopx);
        hipEventSynchronize(stopx);
        hipEventElapsedTime(&time, startx, stopx);
        // cout << loop << ": " << time << " milliseconds" << endl;
        // cout << time << ", " ;

        time_measurements[loop] = time;
        THROW_IF_CUDA_ERROR(hipFree(Forward_InOut_Datas));
        THROW_IF_CUDA_ERROR(hipFree(Root_Table_Device));
        THROW_IF_CUDA_ERROR(hipFree(Inverse_Root_Table_Device));
    }

    cout << endl << endl << "Average: " << calculate_mean(time_measurements, test_count) << endl;
    cout << "Best Average: " << find_min_average(time_measurements, test_count, bestof) << endl;

    cout << "Standart Deviation: " << calculate_standard_deviation(time_measurements, test_count) << endl;

    return EXIT_SUCCESS;
}

/*
cmake . -D CMAKE_CUDA_ARCHITECTURES=86 -D CMAKE_CUDA_COMPILER=$(which nvcc) -D
RING_SIZE=12 -D BATCH_COUNT=4 -B./cmake-build-debug cmake --build
./cmake-build-debug/     --parallel     --target compOpGPU2_bench
./cmake-build-debug/compOpGPU2_bench
*/

/*
// tests

cmake . -D CMAKE_CUDA_ARCHITECTURES=86 -D COPLEX_DATA_TYPE=0
-B./cmake-build-debug cmake --build ./cmake-build-debug/ --target fft_bench
--parallel

./cmake-build-debug/fft_bench 12 16

*/
