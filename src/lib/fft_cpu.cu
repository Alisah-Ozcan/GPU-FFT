#include "hip/hip_runtime.h"
// Copyright 2023-2025 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "fft_cpu.cuh"

namespace gpufft
{

    int bitreverse(int index, int n_power)
    {
        int res_1 = 0;
        for (int i = 0; i < n_power; i++)
        {
            res_1 <<= 1;
            res_1 = (index & 1) | res_1;
            index >>= 1;
        }
        return res_1;
    }

    template <typename T>
    std::vector<T> schoolbook_poly_multiplication(std::vector<T> a,
                                                  std::vector<T> b, T modulus,
                                                  int size)
    {
        std::vector<T> mult_vector(size * 2, 0);

        for (int i = 0; i < size; i++)
        {
            for (int j = 0; j < size; j++)
            {
                T mult = a[i] * b[j];
                mult_vector[i + j] += mult;
            }
        }

        // for(int i = 0; i < 2 * size; i++)
        //{
        //     mult_vector[i] = mult_vector[i] % modulus;
        // }

        for (int i = 0; i < size; i++)
        {
            mult_vector[i] = (mult_vector[i] - mult_vector[i + size]) % modulus;
        }

        return mult_vector;
    }

    template <typename T>
    std::vector<T> schoolbook_poly_multiplication_without_reduction(
        std::vector<T> a, std::vector<T> b, T modulus, int size)
    {
        std::vector<T> mult_vector(size * 2, 0);

        for (int i = 0; i < size; i++)
        {
            for (int j = 0; j < size; j++)
            {
                T mult = a[i] * b[j];
                mult_vector[i + j] += mult;
            }
        }

        for (int i = 0; i < 2 * size; i++)
        {
            mult_vector[i] = mult_vector[i] % modulus;
        }

        return mult_vector;
    }

    template std::vector<int> schoolbook_poly_multiplication(std::vector<int> a,
                                                             std::vector<int> b,
                                                             int modulus,
                                                             int size);

    template std::vector<unsigned long long>
    schoolbook_poly_multiplication(std::vector<unsigned long long> a,
                                   std::vector<unsigned long long> b,
                                   unsigned long long modulus, int size);

    template std::vector<int> schoolbook_poly_multiplication_without_reduction(
        std::vector<int> a, std::vector<int> b, int modulus, int size);

    template std::vector<unsigned long long>
    schoolbook_poly_multiplication_without_reduction(
        std::vector<unsigned long long> a, std::vector<unsigned long long> b,
        unsigned long long modulus, int size);

    template <typename T> FFT<T>::FFT(int size)
    {
        n = size;
        logn = int(log2(n));
        COMPLEX<T> x_(1.0, 0.0);
        x = x_;
        max_size = n * 2;
        root = COMPLEX<T>(2.0) * COMPLEX<T>(M_PI) /
               COMPLEX<T>(max_size); // it was float
        n_inverse = 1.0 / max_size;

        GenerateRootTable();
        GenerateInverseRootTable();
    }

    template <typename T> void FFT<T>::GenerateRootTable()
    {
        COMPLEX<T> j(0.0, 1.0); // Define the complex unit (imaginary part)

        for (int i = 0; i < n; i++)
        {
            COMPLEX<T> element = complex_arithmetic::exp(j * COMPLEX<T>(i) *
                                                         root); // it was float
            root_tables.push_back(element);
        }
    }

    template <typename T> void FFT<T>::GenerateInverseRootTable()
    {
        COMPLEX<T> one(1.0); // Define the complex unit (imaginary part)

        for (int i = 0; i < n; i++)
        {
            COMPLEX<T> element = one / root_tables[i];
            inverse_root_tables.push_back(element);
        }
    }

    template <typename T> void FFT<T>::fft(std::vector<COMPLEX<T>>& input)
    {
        int t = max_size;
        int m = 1;

        while (m < max_size)
        {
            t = t >> 1;

            for (int i = 0; i < m; i++)
            {
                int j1 = 2 * i * t;
                int j2 = j1 + t - 1;

                int index = bitreverse(i, logn);

                COMPLEX<T> S = root_tables[index];

                for (int j = j1; j < (j2 + 1); j++)
                {
                    COMPLEX<T> U = input[j];
                    COMPLEX<T> V = input[j + t];

                    input[j] = U + (V * S);
                    input[j + t] = U - (V * S);
                }
            }

            m = m << 1;
        }
    }

    template <typename T> void FFT<T>::ifft(std::vector<COMPLEX<T>>& input)
    {
        int t = 1;
        int m = max_size;
        while (m > 1)
        {
            int j1 = 0;
            int h = m >> 1;
            for (int i = 0; i < h; i++)
            {
                int j2 = j1 + t - 1;
                int index = bitreverse(i, logn);

                COMPLEX<T> S = inverse_root_tables[index];

                for (int j = j1; j < (j2 + 1); j++)
                {
                    COMPLEX<T> U = input[j];
                    COMPLEX<T> V = input[j + t];

                    input[j] = (U + V);
                    input[j + t] = (U - V) * S;
                }

                j1 = j1 + (t << 1);
            }

            t = t << 1;
            m = m >> 1;
        }

        for (int i = 0; i < max_size; i++)
        {
            input[i] = input[i] * n_inverse;
        }
    }

    template <typename T> std::vector<COMPLEX<T>> FFT<T>::ReverseRootTable()
    {
        std::vector<COMPLEX<T>> reverse_root_table;

        int lg = log2(n);
        for (int i = 0; i < n; i++)
        {
            reverse_root_table.push_back(root_tables[bitreverse(i, lg)]);
        }

        return reverse_root_table;
    }

    template <typename T>
    std::vector<COMPLEX<T>> FFT<T>::InverseReverseRootTable()
    {
        std::vector<COMPLEX<T>> inverse_reverse_root_table;

        int lg = log2(n);
        for (int i = 0; i < n; i++)
        {
            inverse_reverse_root_table.push_back(
                inverse_root_tables[bitreverse(i, lg)]);
        }

        return inverse_reverse_root_table;
    }

    template class FFT<Float32>;
    template class FFT<Float64>;

    ////////////////////////////////////////////////
    ////////////////////////////////////////////////

    template <typename T> FFNT<T>::FFNT(int size)
    {
        n = size;
        logn = int(log2(n));
        COMPLEX<T> x_(1.0, 0.0);
        x = x_;
        max_size = n * 2;
        root = 2.0 * M_PI / static_cast<T>(max_size);
        n_inverse = 1.0 / max_size;

        GenerateRootTable();
        GenerateInverseRootTable();

        GenerateRootTableTwist();
        GenerateInverseRootTableTwist();
    }

    template <typename T> void FFNT<T>::GenerateRootTable()
    {
        COMPLEX<T> j(0.0, 1.0); // Define the complex unit (imaginary part)

        T root_new = 2.0 * M_PI / static_cast<T>(n >> 1);

        for (int i = 0; i < (n >> 1); i++)
        {
            COMPLEX<T> element = complex_arithmetic::exp(j * COMPLEX<T>(i) *
                                                         COMPLEX<T>(root_new));
            root_tables_new.push_back(element);
        }
    }

    template <typename T> void FFNT<T>::GenerateInverseRootTable()
    {
        COMPLEX<T> one(1.0); // Define the complex unit (imaginary part)

        for (int i = 0; i < (n >> 1); i++)
        {
            COMPLEX<T> element = one / root_tables_new[i];
            inverse_root_tables_new.push_back(element);
        }
    }

    template <typename T> void FFNT<T>::GenerateRootTableTwist()
    {
        COMPLEX<T> j(0.0, 1.0); // Define the complex unit (imaginary part)

        T root_new = 2.0 * M_PI / static_cast<T>(2 * n);

        for (int i = 0; i < (n >> 1); i++)
        {
            COMPLEX<T> element = complex_arithmetic::exp(j * COMPLEX<T>(i) *
                                                         COMPLEX<T>(root_new));
            root_tables_twist.push_back(element);
        }
    }

    template <typename T> void FFNT<T>::GenerateInverseRootTableTwist()
    {
        COMPLEX<T> one(1.0); // Define the complex unit (imaginary part)

        for (int i = 0; i < (n >> 1); i++)
        {
            COMPLEX<T> element = one / root_tables_twist[i];
            inverse_root_tables_twist.push_back(element);
        }
    }

    template <typename T>
    std::vector<unsigned long long>
    FFNT<T>::MULT(std::vector<unsigned long long>& input1,
                  std::vector<unsigned long long>& input2)
    {
        std::vector<COMPLEX<T>> input1_complex;
        std::vector<COMPLEX<T>> input2_complex;
        for (int i = 0; i < (n >> 1); i++)
        {
            COMPLEX<T> temp1(input1[i], input1[i + (n >> 1)]); // fold
            temp1 = temp1 * root_tables_twist[i];
            input1_complex.push_back(temp1);

            COMPLEX<T> temp2(input2[i], input2[i + (n >> 1)]); // fold
            temp2 = temp2 * root_tables_twist[i];
            input2_complex.push_back(temp2);
        }

        // FFT1
        int t = (n >> 1);
        int logn_new = int(log2((n >> 1)));
        int m = 1;

        while (m < (n >> 1))
        {
            t = t >> 1;

            for (int i = 0; i < m; i++)
            {
                int j1 = 2 * i * t;
                int j2 = j1 + t - 1;

                int index = bitreverse(i, logn_new - 1);

                COMPLEX<T> S = root_tables_new[index];

                for (int j = j1; j < (j2 + 1); j++)
                {
                    COMPLEX<T> U = input1_complex[j];
                    COMPLEX<T> V = input1_complex[j + t];

                    input1_complex[j] = U + (V * S);
                    input1_complex[j + t] = U - (V * S);
                }
            }

            m = m << 1;
        }

        // FFT2
        t = (n >> 1);
        logn_new = int(log2((n >> 1)));
        m = 1;

        while (m < (n >> 1))
        {
            t = t >> 1;

            for (int i = 0; i < m; i++)
            {
                int j1 = 2 * i * t;
                int j2 = j1 + t - 1;

                int index = bitreverse(i, logn_new - 1);

                COMPLEX<T> S = root_tables_new[index];

                for (int j = j1; j < (j2 + 1); j++)
                {
                    COMPLEX<T> U = input2_complex[j];
                    COMPLEX<T> V = input2_complex[j + t];

                    input2_complex[j] = U + (V * S);
                    input2_complex[j + t] = U - (V * S);
                }
            }

            m = m << 1;
        }

        // Multiplication
        for (int i = 0; i < (n >> 1); i++)
        {
            input1_complex[i] = input1_complex[i] * input2_complex[i];
        }

        // IFFT
        t = 1;
        m = (n >> 1);
        while (m > 1)
        {
            int j1 = 0;
            int h = m >> 1;
            for (int i = 0; i < h; i++)
            {
                int j2 = j1 + t - 1;
                int index = bitreverse(i, logn_new - 1);

                COMPLEX<T> S = inverse_root_tables_new[index];

                for (int j = j1; j < (j2 + 1); j++)
                {
                    COMPLEX<T> U = input1_complex[j];
                    COMPLEX<T> V = input1_complex[j + t];

                    input1_complex[j] = (U + V);
                    input1_complex[j + t] = (U - V) * S;
                }

                j1 = j1 + (t << 1);
            }

            t = t << 1;
            m = m >> 1;
        }

        T n_inverse_new = 1.0 / (n >> 1);
        std::vector<unsigned long long> result(n);
        for (int i = 0; i < (n >> 1); i++)
        {
            input1_complex[i] =
                input1_complex[i] * inverse_root_tables_twist[i]; // untwist
            input1_complex[i] =
                input1_complex[i] * COMPLEX<T>(n_inverse_new); // div

            result[i] = std::round(input1_complex[i].real()); // unfold
            result[i + (n >> 1)] =
                std::round(input1_complex[i].imag()); // unfold
        }

        return result;
    }

    template <typename T>
    std::vector<COMPLEX<T>> FFNT<T>::ReverseRootTable_ffnt()
    {
        std::vector<COMPLEX<T>> reverse_root_table;
        int logn_new = int(log2((n >> 1)));
        for (int i = 0; i < (n >> 1); i++)
        {
            reverse_root_table.push_back(
                root_tables_new[bitreverse(i, logn_new - 1)]);
        }

        return reverse_root_table;
    }

    template <typename T>
    std::vector<COMPLEX<T>> FFNT<T>::InverseReverseRootTable_ffnt()
    {
        std::vector<COMPLEX<T>> inverse_reverse_root_table;
        int logn_new = int(log2((n >> 1)));
        for (int i = 0; i < (n >> 1); i++)
        {
            inverse_reverse_root_table.push_back(
                inverse_root_tables_new[bitreverse(i, logn_new - 1)]);
        }

        return inverse_reverse_root_table;
    }

    template <typename T> std::vector<COMPLEX<T>> FFNT<T>::twist_table_ffnt()
    {
        return root_tables_twist;
    }

    template <typename T> std::vector<COMPLEX<T>> FFNT<T>::untwist_table_ffnt()
    {
        return inverse_root_tables_twist;
    }

    template class FFNT<Float32>;
    template class FFNT<Float64>;

} // namespace gpufft
