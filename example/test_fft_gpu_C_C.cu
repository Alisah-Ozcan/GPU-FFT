#include "hip/hip_runtime.h"
// Copyright 2023-2025 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include <cstdlib> // For atoi or atof functions
#include <iomanip>
#include <iostream>
#include <random>

#include "fft.cuh"
#include "fft_cpu.cuh"

using namespace std;
using namespace gpufft;

int q;
int logn;
int batch;
int n;

// typedef Float32 TestDataType; // Use for 32-bit benchmark
typedef Float64 TestDataType; // Use for 64-bit benchmark

int main(int argc, char* argv[])
{
    if (argc < 3)
    {
        q = 7;
        logn = 12;
        batch = 1;
        n = 1 << logn;
    }
    else
    {
        q = 7;
        logn = atoi(argv[1]);
        batch = atoi(argv[2]);
        n = 1 << logn;

        if ((logn < 12) || (24 < logn))
        {
            throw std::runtime_error("LOGN should be in range 12 to 24.");
        }
    }

    std::random_device rd;
    std::mt19937 gen(rd());
    int minNumber = 0;
    int maxNumber = q - 1;
    std::uniform_int_distribution<int> dis(minNumber, maxNumber);

    vector<vector<int>> A_poly(batch, vector<int>(n * 2));
    vector<vector<int>> B_poly(batch, vector<int>(n * 2));

    for (int j = 0; j < batch; j++)
    {
        for (int i = 0; i < n; i++)
        {
            A_poly[j][i] = dis(gen);
            B_poly[j][i] = dis(gen);
        }
    }

    // Zero Pad
    for (int j = 0; j < batch; j++)
    {
        for (int i = 0; i < n; i++)
        {
            A_poly[j][i + n] = 0;
            B_poly[j][i + n] = 0;
        }
    }

    std::vector<std::vector<COMPLEX<TestDataType>>> A_vec(
        batch, std::vector<COMPLEX<TestDataType>>(n * 2));
    std::vector<std::vector<COMPLEX<TestDataType>>> B_vec(
        batch, std::vector<COMPLEX<TestDataType>>(n * 2));

    std::vector<std::vector<COMPLEX<TestDataType>>> vec_GPU(
        2 * batch,
        std::vector<COMPLEX<TestDataType>>(n * 2)); // A and B together

    for (int j = 0; j < batch; j++)
    {
        for (int i = 0; i < n * 2; i++)
        {
            COMPLEX<TestDataType> A_element = A_poly[j][i];
            A_vec[j][i] = A_element;

            COMPLEX<TestDataType> B_element = B_poly[j][i];
            B_vec[j][i] = B_element;
        }
    }

    for (int j = 0; j < batch; j++)
    { // LOAD A
        for (int i = 0; i < n * 2; i++)
        {
            COMPLEX<TestDataType> element = A_poly[j][i];
            vec_GPU[j][i] = element;
        }
    }

    for (int j = 0; j < batch; j++)
    { // LOAD B
        for (int i = 0; i < n * 2; i++)
        {
            COMPLEX<TestDataType> element = B_poly[j][i];
            vec_GPU[j + batch][i] = element;
        }
    }

    FFT<TestDataType> fft_generator(n);

    /////////////////////////////////////////////////////////////////////////

    COMPLEX<TestDataType>* Forward_InOut_Datas;

    GPUFFT_CUDA_CHECK(hipMalloc(
        &Forward_InOut_Datas,
        2 * batch * n * 2 *
            sizeof(COMPLEX<TestDataType>))); // 2 --> A and B, batch -->
                                             // batch size, 2 --> zero pad

    for (int j = 0; j < 2 * batch; j++)
    {
        GPUFFT_CUDA_CHECK(hipMemcpy(
            Forward_InOut_Datas + (n * 2 * j), vec_GPU[j].data(),
            n * 2 * sizeof(COMPLEX<TestDataType>), hipMemcpyHostToDevice));
    }
    /////////////////////////////////////////////////////////////////////////

    COMPLEX<TestDataType>* Root_Table_Device;

    GPUFFT_CUDA_CHECK(
        hipMalloc(&Root_Table_Device, n * sizeof(COMPLEX<TestDataType>)));

    vector<COMPLEX<TestDataType>> reverse_table =
        fft_generator.ReverseRootTable();
    GPUFFT_CUDA_CHECK(hipMemcpy(Root_Table_Device, reverse_table.data(),
                                 n * sizeof(COMPLEX<TestDataType>),
                                 hipMemcpyHostToDevice));

    /////////////////////////////////////////////////////////////////////////

    COMPLEX<TestDataType>* Inverse_Root_Table_Device;

    GPUFFT_CUDA_CHECK(hipMalloc(&Inverse_Root_Table_Device,
                                 n * sizeof(COMPLEX<TestDataType>)));

    vector<COMPLEX<TestDataType>> inverse_reverse_table =
        fft_generator.InverseReverseRootTable();
    GPUFFT_CUDA_CHECK(
        hipMemcpy(Inverse_Root_Table_Device, inverse_reverse_table.data(),
                   n * sizeof(COMPLEX<TestDataType>), hipMemcpyHostToDevice));

    /////////////////////////////////////////////////////////////////////////

    fft_configuration<TestDataType> cfg_fft{};
    cfg_fft.n_power = (logn + 1);
    cfg_fft.fft_type = FORWARD;
    cfg_fft.reduction_poly = ReductionPolynomial::X_N_minus;
    cfg_fft.zero_padding = false;
    cfg_fft.stream = 0;

    GPU_FFT(Forward_InOut_Datas, Root_Table_Device, cfg_fft, batch * 2, false);

    fft_configuration<TestDataType> cfg_ifft{};
    cfg_ifft.n_power = (logn + 1);
    cfg_ifft.fft_type = INVERSE;
    cfg_ifft.reduction_poly = ReductionPolynomial::X_N_minus;
    cfg_ifft.zero_padding = false;
    cfg_ifft.mod_inverse = COMPLEX<TestDataType>(fft_generator.n_inverse, 0.0);
    cfg_ifft.stream = 0;

    GPU_FFT(Forward_InOut_Datas, Inverse_Root_Table_Device, cfg_ifft, batch,
            true);

    COMPLEX<TestDataType> test[batch * 2 * n];
    GPUFFT_CUDA_CHECK(hipMemcpy(test, Forward_InOut_Datas,
                                 batch * n * 2 * sizeof(COMPLEX<TestDataType>),
                                 hipMemcpyDeviceToHost));

    for (int j = 0; j < batch; j++)
    {
        vector<int> test_school =
            schoolbook_poly_multiplication_without_reduction(A_poly[j],
                                                             B_poly[j], q, n);
        for (int i = 0; i < n * 2; i++)
        {
            signed gpu_result = std::round(test[(j * (n * 2)) + i].real());
            if (test_school[i] != (gpu_result % q))
            {
                throw runtime_error("ERROR");
            }

            if (i < 10)
            {
                cout << test_school[i] << " - " << gpu_result % q << endl;
            }
        }
    }

    return EXIT_SUCCESS;
}
