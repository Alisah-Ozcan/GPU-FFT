#include "hip/hip_runtime.h"
// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include <cstdlib> // For atoi or atof functions
#include <iomanip>
#include <iostream>
#include <random>

#include "fft.cuh"
#include "fft_cpu.cuh"

using namespace std;
using namespace fft;

int q;
int logn;
int batch;
int n;

int main(int argc, char* argv[])
{
    if (argc < 3)
    {
        q = 7;
        logn = 12;
        batch = 1;
        n = 1 << logn;
    }
    else
    {
        q = 7;
        logn = atoi(argv[1]);
        batch = atoi(argv[2]);
        n = 1 << logn;

        if ((logn < 12) || (24 < logn))
        {
            throw std::runtime_error("LOGN should be in range 12 to 24.");
        }
    }

    std::random_device rd;
    std::mt19937 gen(rd());
    unsigned long long minNumber = 0;
    unsigned long long maxNumber = q - 1;
    std::uniform_int_distribution<int> dis(minNumber, maxNumber);

    vector<vector<unsigned long long>> A_poly(
        batch, vector<unsigned long long>(n * 2));
    vector<vector<unsigned long long>> B_poly(
        batch, vector<unsigned long long>(n * 2));

    for (int j = 0; j < batch; j++)
    {
        for (int i = 0; i < n; i++)
        {
            A_poly[j][i] = dis(gen);
            B_poly[j][i] = dis(gen);
        }
    }

    // Zero Pad
    for (int j = 0; j < batch; j++)
    {
        for (int i = 0; i < n; i++)
        {
            A_poly[j][i + n] = 0;
            B_poly[j][i + n] = 0;
        }
    }

    std::vector<std::vector<FIXED_POINT>> vec_GPU(
        2 * batch, std::vector<FIXED_POINT>(n * 2)); // A and B together

    for (int j = 0; j < batch; j++)
    { // LOAD A
        for (int i = 0; i < n * 2; i++)
        {
            vec_GPU[j][i] = static_cast<FIXED_POINT>(A_poly[j][i]);
        }
    }

    for (int j = 0; j < batch; j++)
    { // LOAD B
        for (int i = 0; i < n * 2; i++)
        {
            vec_GPU[j + batch][i] = static_cast<FIXED_POINT>(B_poly[j][i]);
        }
    }

    FFT fft_generator(n);

    /////////////////////////////////////////////////////////////////////////

    COMPLEX* Temp_Datas;
    FFT_CUDA_CHECK(hipMalloc(
        &Temp_Datas,
        2 * batch * n * 2 * sizeof(COMPLEX))); // 2 --> A and B, batch -->
                                               // batch size, 2 --> zero pad

    FIXED_POINT* InOut_Datas;
    FFT_CUDA_CHECK(hipMalloc(
        &InOut_Datas,
        2 * batch * n * 2 * sizeof(FIXED_POINT))); // 2 --> A and B, batch -->
                                                   // batch size, 2 --> zero pad

    for (int j = 0; j < 2 * batch; j++)
    {
        FFT_CUDA_CHECK(hipMemcpy(InOut_Datas + (n * 2 * j), vec_GPU[j].data(),
                                  n * 2 * sizeof(FIXED_POINT),
                                  hipMemcpyHostToDevice));
    }
    /////////////////////////////////////////////////////////////////////////

    COMPLEX* Root_Table_Device;

    FFT_CUDA_CHECK(hipMalloc(&Root_Table_Device, n * sizeof(COMPLEX)));

    vector<COMPLEX_C> reverse_table = fft_generator.ReverseRootTable();
    FFT_CUDA_CHECK(hipMemcpy(Root_Table_Device, reverse_table.data(),
                              n * sizeof(COMPLEX), hipMemcpyHostToDevice));

    /////////////////////////////////////////////////////////////////////////

    COMPLEX* Inverse_Root_Table_Device;

    FFT_CUDA_CHECK(hipMalloc(&Inverse_Root_Table_Device, n * sizeof(COMPLEX)));

    vector<COMPLEX_C> inverse_reverse_table =
        fft_generator.InverseReverseRootTable();
    FFT_CUDA_CHECK(hipMemcpy(Inverse_Root_Table_Device,
                              inverse_reverse_table.data(), n * sizeof(COMPLEX),
                              hipMemcpyHostToDevice));

    /////////////////////////////////////////////////////////////////////////

    fft_configuration cfg_fft = {.n_power = (logn + 1),
                                 .ntt_type = FORWARD,
                                 .reduction_poly =
                                     ReductionPolynomial::X_N_minus,
                                 .zero_padding = false,
                                 .stream = 0};
    GPU_FFT(InOut_Datas, Temp_Datas, Root_Table_Device, cfg_fft, batch * 2,
            false);

    fft_configuration cfg_ifft = {
        .n_power = (logn + 1),
        .ntt_type = INVERSE,
        .reduction_poly = ReductionPolynomial::X_N_minus,
        .zero_padding = false,
        .mod_inverse = COMPLEX(fft_generator.n_inverse, 0.0),
        .stream = 0};

    GPU_FFT(InOut_Datas, Temp_Datas, Inverse_Root_Table_Device, cfg_ifft, batch,
            true);

    FIXED_POINT test[batch * 2 * n];
    FFT_CUDA_CHECK(hipMemcpy(test, InOut_Datas,
                              batch * n * 2 * sizeof(FIXED_POINT),
                              hipMemcpyDeviceToHost));

    for (int j = 0; j < batch; j++)
    {
        vector<unsigned long long> test_school =
            schoolbook_poly_multiplication(A_poly[j], B_poly[j], q, n);
        for (int i = 0; i < n * 2; i++)
        {
            signed gpu_result = std::round(test[(j * (n * 2)) + i]);
            if (test_school[i] != (gpu_result % q))
            {
                throw("ERROR");
            }

            if (i < 10)
            {
                cout << test_school[i] << " - " << gpu_result % q << endl;
            }
        }
    }

    return EXIT_SUCCESS;
}
