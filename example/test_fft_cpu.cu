#include "hip/hip_runtime.h"
// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include <cstdlib> // For atoi or atof functions
#include <iomanip>
#include <iostream>
#include <random>

#include "fft.cuh"
#include "fft_cpu.cuh"

using namespace std;
using namespace gpufft;

int q;
int logn;
int batch;
int n;

int main(int argc, char* argv[])
{
    if (argc < 3)
    {
        q = 7;
        logn = 12;
        batch = 1;
        n = 1 << logn;
    }
    else
    {
        q = 7;
        logn = atoi(argv[1]);
        batch = atoi(argv[2]);
        n = 1 << logn;

        if ((logn < 12) || (24 < logn))
        {
            throw std::runtime_error("LOGN should be in range 12 to 24.");
        }
    }

    std::random_device rd;
    std::mt19937 gen(rd());
    unsigned long long minNumber = 0;
    unsigned long long maxNumber = q - 1;
    std::uniform_int_distribution<int> dis(minNumber, maxNumber);

    vector<vector<unsigned long long>> A_poly(
        batch, vector<unsigned long long>(n * 2));
    vector<vector<unsigned long long>> B_poly(
        batch, vector<unsigned long long>(n * 2));

    for (int j = 0; j < batch; j++)
    {
        for (int i = 0; i < n; i++)
        {
            A_poly[j][i] = dis(gen);
            B_poly[j][i] = dis(gen);
        }
    }

    // Zero Pad
    for (int j = 0; j < batch; j++)
    {
        for (int i = 0; i < n; i++)
        {
            A_poly[j][i + n] = 0;
            B_poly[j][i + n] = 0;
        }
    }

    std::vector<std::vector<Complex64>> A_vec(batch,
                                              std::vector<Complex64>(n * 2));
    std::vector<std::vector<Complex64>> B_vec(batch,
                                              std::vector<Complex64>(n * 2));

    std::vector<std::vector<Complex64>> vec_GPU(
        2 * batch, std::vector<Complex64>(n * 2)); // A and B together

    for (int j = 0; j < batch; j++)
    {
        for (int i = 0; i < n * 2; i++)
        {
            Complex64 A_element = A_poly[j][i];
            A_vec[j][i] = A_element;

            Complex64 B_element = B_poly[j][i];
            B_vec[j][i] = B_element;
        }
    }

    for (int j = 0; j < batch; j++)
    { // LOAD A
        for (int i = 0; i < n * 2; i++)
        {
            Complex64 element = A_poly[j][i];
            vec_GPU[j][i] = element;
        }
    }

    for (int j = 0; j < batch; j++)
    { // LOAD B
        for (int i = 0; i < n * 2; i++)
        {
            Complex64 element = B_poly[j][i];
            vec_GPU[j + batch][i] = element;
        }
    }

    FFT<Float64> fft_generator(n);

    for (int j = 0; j < batch; j++)
    {
        fft_generator.fft(A_vec[j]);
        fft_generator.fft(B_vec[j]);
    }

    for (int j = 0; j < batch; j++)
    {
        for (int i = 0; i < n * 2; i++)
        {
            A_vec[j][i] = A_vec[j][i] * B_vec[j][i];
        }
    }

    for (int j = 0; j < batch; j++)
    {
        fft_generator.ifft(A_vec[j]);
    }

    for (int j = 0; j < batch; j++)
    {
        vector<unsigned long long> test_school =
            schoolbook_poly_multiplication_without_reduction(A_poly[j],
                                                             B_poly[j], q, n);
        for (int i = 0; i < n * 2; i++)
        {
            signed cpu_result = std::round(A_vec[j][i].real());

            if (test_school[i] != (cpu_result % q))
            {
                throw("ERROR");
            }

            if (i < 10)
            {
                cout << test_school[i] << " - " << cpu_result % q << endl;
            }
        }
    }

    return EXIT_SUCCESS;
}
